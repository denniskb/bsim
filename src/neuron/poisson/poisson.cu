#include "hip/hip_runtime.h"

#include "../../gpu_utils/runtime.h"

#include "GPoisson.h"


__global__ void update_poisson_neuron(GPoissonNeurons *d_neurons, int const num, int const start_id)
{
	for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num; idx += blockDim.x * gridDim.x) {
		if (hiprand_uniform(&d_neurons->p_state[idx]) < 0.002f) {
			gFireCount[start_id + idx]++;
			gFiredTable[gFiredTableCap*gCurrentIdx + atomicAdd(&(gFiredTableSizes[gCurrentIdx]), 1)] = start_id + idx;
		}
	}
}
